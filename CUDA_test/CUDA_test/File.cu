#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


//CUDA Kernel for vec Addition
__global__ void vecAdd(float* a, float* b, float* c, int num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num)
	{
		c[idx] = a[idx] + b[idx];
	}
}


int main()
{
	int N = 1024;
	float* a, * b, * c;

	for (int i = 0;i < N;i++)
	{
		a[i] = 1.0f;
		b[i] = 2.0f;
	}
	float* a_d, * b_d, * c_d;
	hipMalloc(&a_d, N * sizeof(float));
	hipMalloc(&b_d, N * sizeof(float));
	hipMalloc(&c_d, N * sizeof(float));

	hipMemcpy(a_d, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, N * sizeof(float), hipMemcpyHostToDevice);

	int blockSize = 256;
	int gridSize = (N + blockSize - 1) / blockSize;

	vecAdd << <gridSize, blockSize >> > (a_d, b_d, c_d, N);

	hipMemcpy(a, a_d, N * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0;i < N;i++)
		std::cout << c[i] << " " << std::endl;

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	delete a;
	delete b;
	delete c;

	return 0;

}