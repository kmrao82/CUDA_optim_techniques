#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void vecAdd(float* a, float* b, float* c, int num) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 1024;
    std::vector<float> a(N), b(N), c(N); // Resized to N

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    float* a_d, * b_d, * c_d;
    hipMalloc(&a_d, N * sizeof(float));
    hipMalloc(&b_d, N * sizeof(float));
    hipMalloc(&c_d, N * sizeof(float));

    hipMemcpy(a_d, a.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    int gridSize = (N + maxThreadsPerBlock - 1) / maxThreadsPerBlock;

    std::cout << "Kernel params: Threads per block " << maxThreadsPerBlock
        << " grid size " << gridSize << std::endl;

    vecAdd <<<gridSize, maxThreadsPerBlock >>> (a_d, b_d, c_d, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipMemcpy(c.data(), c_d, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        std::cout << c[i] << " ";
    }

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return 0;
}
